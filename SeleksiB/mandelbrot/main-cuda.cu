#include "hip/hip_runtime.h"
#include <SFML/Graphics.hpp>
#include <iostream>
#include <complex>
#include <vector>
#include <chrono>
#include <thread>
#include <cmath>
#include <sstream>
#include <future>
#include <atomic>
#include <mutex>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <>

class Button {
private:
    sf::RectangleShape shape;
    sf::Text text;
    sf::Font* font;
    bool is_pressed;
    
public:
    Button(float x, float y, float width, float height, const std::string& label, sf::Font& f) {
        font = &f;
        shape.setPosition(x, y);
        shape.setSize(sf::Vector2f(width, height));
        shape.setFillColor(sf::Color(70, 70, 70));
        shape.setOutlineColor(sf::Color(150, 150, 150));
        shape.setOutlineThickness(2);
        
        text.setFont(f);
        text.setString(label);
        text.setCharacterSize(14);
        text.setFillColor(sf::Color::White);
        
        sf::FloatRect textBounds = text.getLocalBounds();
        text.setPosition(
            x + (width - textBounds.width) / 2,
            y + (height - textBounds.height) / 2 - 2
        );
        
        is_pressed = false;
    }
    
    bool isClicked(sf::Vector2i mousePos) {
        return shape.getGlobalBounds().contains(static_cast<float>(mousePos.x), static_cast<float>(mousePos.y));
    }
    
    void setPressed(bool pressed) {
        is_pressed = pressed;
        if (pressed) {
            shape.setFillColor(sf::Color(100, 100, 100));
        } else {
            shape.setFillColor(sf::Color(70, 70, 70));
        }
    }
    
    void draw(sf::RenderWindow& window) {
        window.draw(shape);
        window.draw(text);
    }
    
    void updateText(const std::string& newText) {
        text.setString(newText);
        sf::FloatRect textBounds = text.getLocalBounds();
        sf::Vector2f pos = shape.getPosition();
        sf::Vector2f size = shape.getSize();
        text.setPosition(
            pos.x + (size.x - textBounds.width) / 2,
            pos.y + (size.y - textBounds.height) / 2 - 2
        );
    }
};

__device__ int cuda_mandelbrotIteration(double real, double imag, int max_iterations) {
    double z_real = 0.0;
    double z_imag = 0.0;
    int n = 0;
    
    while (z_real * z_real + z_imag * z_imag <= 4.0 && n < max_iterations) {
        double temp = z_real * z_real - z_imag * z_imag + real;
        z_imag = 2.0 * z_real * z_imag + imag;
        z_real = temp;
        n++;
    }
    
    return n;
}

__device__ int cuda_juliaIteration(double z_real, double z_imag, double c_real, double c_imag, int max_iterations) {
    int n = 0;
    
    while (z_real * z_real + z_imag * z_imag <= 4.0 && n < max_iterations) {
        double temp = z_real * z_real - z_imag * z_imag + c_real;
        z_imag = 2.0 * z_real * z_imag + c_imag;
        z_real = temp;
        n++;
    }
    
    return n;
}

__device__ void cuda_getColor(int iterations, int max_iterations, unsigned char* r, unsigned char* g, unsigned char* b) {
    if (iterations == max_iterations) {
        *r = *g = *b = 0;
        return;
    }
    
    double t = (double)iterations / max_iterations;
    
    *r = (unsigned char)(255 * (0.5 + 0.5 * cos(3.0 + t * 6.28)));
    *g = (unsigned char)(255 * (0.5 + 0.5 * cos(2.0 + t * 6.28)));
    *b = (unsigned char)(255 * (0.5 + 0.5 * cos(1.0 + t * 6.28)));
}

__global__ void cuda_generateFractal(unsigned char* pixels, int width, int height, 
                                    double min_real, double max_real, double min_imag, double max_imag,
                                    int max_iterations, bool julia_mode, double julia_c_real, double julia_c_imag) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
    
    double real = min_real + (max_real - min_real) * x / (width - 1);
    double imag = min_imag + (max_imag - min_imag) * y / (height - 1);
    
    int iterations;
    if (julia_mode) {
        iterations = cuda_juliaIteration(real, imag, julia_c_real, julia_c_imag, max_iterations);
    } else {
        iterations = cuda_mandelbrotIteration(real, imag, max_iterations);
    }
    
    unsigned char r, g, b;
    cuda_getColor(iterations, max_iterations, &r, &g, &b);
    
    int index = (y * width + x) * 4;
    pixels[index] = r;         
    pixels[index + 1] = g;     
    pixels[index + 2] = b;     
    pixels[index + 3] = 255;   
}

class MandelbrotViewer {
private:
    int fractal_width, fractal_height, window_width, window_height;
    int max_iterations;
    double min_real, max_real, min_imag, max_imag;
    sf::RenderWindow window;
    sf::Image image;
    sf::Texture texture;
    sf::Sprite sprite;
    std::vector<sf::Uint8> pixels;
    sf::Font font;
    
    bool julia_mode;
    std::complex<double> julia_c;
    
    bool left_dragging;
    sf::Vector2i drag_start;
    sf::Vector2i current_mouse_pos;
    
    std::vector<Button> buttons;
    sf::Text info_text;
    sf::Text cursor_text;
    
    sf::Clock julia_update_clock;
    const float julia_update_interval = 0.1f;
    
    std::atomic<bool> is_generating;
    std::mutex pixel_mutex;
    int num_threads;
    
    unsigned char* d_pixels;
    bool cuda_available;
    bool use_cuda;
    
    enum ComputeMode { CPU_SERIAL, CPU_PARALLEL, GPU_CUDA };
    std::vector<double> benchmark_times;

public:
    MandelbrotViewer(int fw, int fh) : fractal_width(fw), fractal_height(fh), max_iterations(100),
                                      julia_mode(false), left_dragging(false), is_generating(false),
                                      d_pixels(nullptr), cuda_available(false), use_cuda(false) {
        initCuda();
        
        num_threads = std::max(1u, std::thread::hardware_concurrency());
        
        int gui_width = 250;
        window_width = fractal_width + gui_width;
        window_height = fractal_height;
        
        min_real = -2.5;
        max_real = 1.0;
        min_imag = -1.25;
        max_imag = 1.25;
        
        julia_c = std::complex<double>(-0.7, 0.27015);
        
        window.create(sf::VideoMode(window_width, window_height), "Interactive Mandelbrot/Julia Set with CUDA");
        window.setFramerateLimit(60);
        
        if (!font.loadFromFile("arial.ttf")) {
            if (!font.loadFromFile("C:/Windows/Fonts/arial.ttf") && 
                !font.loadFromFile("/System/Library/Fonts/Arial.ttf") &&
                !font.loadFromFile("/usr/share/fonts/truetype/dejavu/DejaVuSans.ttf")) {
            }
        }
        
        image.create(fractal_width, fractal_height);
        texture.create(fractal_width, fractal_height);
        pixels.resize(fractal_width * fractal_height * 4);
        
        setupGUI(gui_width);
        generateFractal();
        updateTexture();
    }
    
    ~MandelbrotViewer() {
        if (d_pixels) {
            hipFree(d_pixels);
        }
    }
    
    void initCuda() {
        int deviceCount = 0;
        hipError_t error = hipGetDeviceCount(&deviceCount);
        
        if (error != hipSuccess || deviceCount == 0) {
            cuda_available = false;
            return;
        }
        
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        
        size_t pixel_size = fractal_width * fractal_height * 4 * sizeof(unsigned char);
        error = hipMalloc(&d_pixels, pixel_size);
        
        if (error != hipSuccess) {
            cuda_available = false;
            return;
        }
        
        cuda_available = true;
        use_cuda = true;
    }
    
    void setupGUI(int gui_width) {
        float button_width = gui_width - 20;
        float button_height = 30;
        float start_x = fractal_width + 10;
        float start_y = 20;
        float spacing = button_height + 10;
        
        buttons.clear();
        
        buttons.emplace_back(start_x, start_y, button_width, button_height, 
                           julia_mode ? "Switch to Mandelbrot" : "Switch to Julia", font);
        
        buttons.emplace_back(start_x, start_y + spacing, button_width, button_height, 
                           "CPU Serial", font);
        buttons.emplace_back(start_x, start_y + 2*spacing, button_width, button_height, 
                           "CPU Parallel", font);
        if (cuda_available) {
            buttons.emplace_back(start_x, start_y + 3*spacing, button_width, button_height, 
                               "GPU CUDA (Active)", font);
        }
        
        int offset = cuda_available ? 1 : 0;
        buttons.emplace_back(start_x, start_y + (3+offset)*spacing, button_width, button_height, 
                           "Zoom In (Center)", font);
        buttons.emplace_back(start_x, start_y + (4+offset)*spacing, button_width, button_height, 
                           "Zoom Out (Center)", font);
        
        buttons.emplace_back(start_x, start_y + (5+offset)*spacing, button_width, button_height, 
                           "Reset View", font);
        
        buttons.emplace_back(start_x, start_y + (6+offset)*spacing, button_width, button_height, 
                           "More Iterations (+50)", font);
        buttons.emplace_back(start_x, start_y + (7+offset)*spacing, button_width, button_height, 
                           "Less Iterations (-50)", font);
        
        buttons.emplace_back(start_x, start_y + (8+offset)*spacing, button_width, button_height, 
                           "Run Benchmark", font);
        
        buttons.emplace_back(start_x, start_y + (9+offset)*spacing, button_width, button_height, 
                           "Julia: Classic", font);
        buttons.emplace_back(start_x, start_y + (10+offset)*spacing, button_width, button_height, 
                           "Julia: Dragon", font);
        buttons.emplace_back(start_x, start_y + (11+offset)*spacing, button_width, button_height, 
                           "Julia: Spiral", font);
        
        info_text.setFont(font);
        info_text.setCharacterSize(12);
        info_text.setFillColor(sf::Color::White);
        info_text.setPosition(start_x, start_y + (13+offset)*spacing);
        
        cursor_text.setFont(font);
        cursor_text.setCharacterSize(11);
        cursor_text.setFillColor(sf::Color(200, 200, 200));
        cursor_text.setPosition(start_x, window_height - 60);
        
        updateInfoText();
    }
    
    void updateInfoText() {
        std::stringstream ss;
        ss << "Mode: " << (julia_mode ? "Julia" : "Mandelbrot") << "\n";
        ss << "Compute: ";
        if (use_cuda && cuda_available) {
            ss << "GPU CUDA";
        } else {
            ss << "CPU (" << num_threads << " threads)";
        }
        ss << "\n";
        ss << "Iterations: " << max_iterations << "\n";
        ss << "Zoom: " << std::fixed << std::setprecision(2) << (3.5 / (max_real - min_real)) << "x\n";
        if (julia_mode) {
            ss << "Julia C: " << std::fixed << std::setprecision(3) 
               << julia_c.real() << " + " << julia_c.imag() << "i\n";
        }
        
        if (!benchmark_times.empty() && benchmark_times.size() >= 2) {
            ss << "\nBenchmark Results:\n";
            ss << "CPU Serial: " << std::fixed << std::setprecision(1) << benchmark_times[0] << "ms\n";
            ss << "CPU Parallel: " << std::fixed << std::setprecision(1) << benchmark_times[1] << "ms\n";
            if (cuda_available && benchmark_times.size() >= 3) {
                ss << "GPU CUDA: " << std::fixed << std::setprecision(1) << benchmark_times[2] << "ms\n";
                ss << "Speedup vs Serial: " << std::fixed << std::setprecision(1) 
                   << benchmark_times[0] / benchmark_times[2] << "x\n";
                ss << "Speedup vs Parallel: " << std::fixed << std::setprecision(1) 
                   << benchmark_times[1] / benchmark_times[2] << "x\n";
            }
        }
        
        ss << "\nControls:\n";
        ss << "- Left click + drag to pan\n";
        ss << "- Mouse wheel to zoom\n  at cursor position\n";
        if (julia_mode) {
            ss << "- Move mouse over\n  fractal to change C";
        }
        
        info_text.setString(ss.str());
    }
    
    void updateCursorText() {
        if (current_mouse_pos.x < fractal_width && current_mouse_pos.y >= 0 && 
            current_mouse_pos.y < fractal_height) {
            double real = min_real + (max_real - min_real) * current_mouse_pos.x / (fractal_width - 1);
            double imag = min_imag + (max_imag - min_imag) * current_mouse_pos.y / (fractal_height - 1);
            
            std::stringstream ss;
            ss << "Cursor Position:\n";
            ss << "Screen: (" << current_mouse_pos.x << ", " << current_mouse_pos.y << ")\n";
            ss << "Complex: " << std::fixed << std::setprecision(6) 
               << real << " + " << imag << "i";
            
            cursor_text.setString(ss.str());
        } else {
            cursor_text.setString("Cursor Position:\n(Outside fractal area)");
        }
    }
    
    int mandelbrotIteration(std::complex<double> c) {
        std::complex<double> z = 0;
        int n = 0;
        
        while (std::abs(z) <= 2.0 && n < max_iterations) {
            z = z * z + c;
            n++;
        }
        
        return n;
    }
    
    int juliaIteration(std::complex<double> z, std::complex<double> c) {
        int n = 0;
        
        while (std::abs(z) <= 2.0 && n < max_iterations) {
            z = z * z + c;
            n++;
        }
        
        return n;
    }
    
    sf::Color getColor(int iterations) {
        if (iterations == max_iterations) {
            return sf::Color::Black;
        }
        
        double t = (double)iterations / max_iterations;
        
        uint8_t r = static_cast<uint8_t>(255 * (0.5 + 0.5 * std::cos(3.0 + t * 6.28)));
        uint8_t g = static_cast<uint8_t>(255 * (0.5 + 0.5 * std::cos(2.0 + t * 6.28)));
        uint8_t b = static_cast<uint8_t>(255 * (0.5 + 0.5 * std::cos(1.0 + t * 6.28)));
        
        return sf::Color(r, g, b);
    }
    
    void generateFractalSerial() {
        for (int y = 0; y < fractal_height; y++) {
            for (int x = 0; x < fractal_width; x++) {
                double real = min_real + (max_real - min_real) * x / (fractal_width - 1);
                double imag = min_imag + (max_imag - min_imag) * y / (fractal_height - 1);
                
                int iterations;
                if (julia_mode) {
                    std::complex<double> z(real, imag);
                    iterations = juliaIteration(z, julia_c);
                } else {
                    std::complex<double> c(real, imag);
                    iterations = mandelbrotIteration(c);
                }
                
                sf::Color color = getColor(iterations);
                
                int index = (y * fractal_width + x) * 4;
                pixels[index] = color.r;     
                pixels[index + 1] = color.g; 
                pixels[index + 2] = color.b; 
                pixels[index + 3] = 255;     
            }
        }
    }
    
    void generateFractalChunk(int start_y, int end_y, std::vector<sf::Uint8>& local_pixels) {
        for (int y = start_y; y < end_y; y++) {
            for (int x = 0; x < fractal_width; x++) {
                double real = min_real + (max_real - min_real) * x / (fractal_width - 1);
                double imag = min_imag + (max_imag - min_imag) * y / (fractal_height - 1);
                
                int iterations;
                if (julia_mode) {
                    std::complex<double> z(real, imag);
                    iterations = juliaIteration(z, julia_c);
                } else {
                    std::complex<double> c(real, imag);
                    iterations = mandelbrotIteration(c);
                }
                
                sf::Color color = getColor(iterations);
                
                int index = ((y - start_y) * fractal_width + x) * 4;
                local_pixels[index] = color.r;     
                local_pixels[index + 1] = color.g; 
                local_pixels[index + 2] = color.b; 
                local_pixels[index + 3] = 255;     
            }
        }
    }
    
    void generateFractalParallel() {
        int rows_per_thread = fractal_height / num_threads;
        int remaining_rows = fractal_height % num_threads;
        
        std::vector<std::future<void>> futures;
        std::vector<std::vector<sf::Uint8>> thread_pixels(num_threads);
        
        int current_y = 0;
        for (int t = 0; t < num_threads; t++) {
            int start_y = current_y;
            int rows_to_process = rows_per_thread + (t < remaining_rows ? 1 : 0);
            int end_y = start_y + rows_to_process;
            current_y = end_y;
            
            thread_pixels[t].resize(rows_to_process * fractal_width * 4);
            
            futures.push_back(std::async(std::launch::async, 
                [this, start_y, end_y, &thread_pixels, t]() {
                    generateFractalChunk(start_y, end_y, thread_pixels[t]);
                }));
        }
        
        for (auto& future : futures) {
            future.wait();
        }
        
        current_y = 0;
        for (int t = 0; t < num_threads; t++) {
            int start_y = current_y;
            int rows_to_process = rows_per_thread + (t < remaining_rows ? 1 : 0);
            current_y += rows_to_process;
            
            for (int local_y = 0; local_y < rows_to_process; local_y++) {
                int global_y = start_y + local_y;
                for (int x = 0; x < fractal_width; x++) {
                    int local_index = (local_y * fractal_width + x) * 4;
                    int global_index = (global_y * fractal_width + x) * 4;
                    
                    pixels[global_index] = thread_pixels[t][local_index];         
                    pixels[global_index + 1] = thread_pixels[t][local_index + 1]; 
                    pixels[global_index + 2] = thread_pixels[t][local_index + 2]; 
                    pixels[global_index + 3] = thread_pixels[t][local_index + 3]; 
                }
            }
        }
    }
    
    void generateFractalCuda() {
        if (!cuda_available || !d_pixels) return;
        
        dim3 blockSize(16, 16);
        dim3 gridSize((fractal_width + blockSize.x - 1) / blockSize.x,
                      (fractal_height + blockSize.y - 1) / blockSize.y);
        
        cuda_generateFractal<<<gridSize, blockSize>>>(
            d_pixels, fractal_width, fractal_height,
            min_real, max_real, min_imag, max_imag,
            max_iterations, julia_mode, julia_c.real(), julia_c.imag()
        );
        
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            return;
        }
        
        hipDeviceSynchronize();
        
        size_t pixel_size = fractal_width * fractal_height * 4 * sizeof(unsigned char);
        hipMemcpy(pixels.data(), d_pixels, pixel_size, hipMemcpyDeviceToHost);
    }
    
    void generateFractal() {
        if (is_generating.exchange(true)) {
            return;
        }
        
        auto start = std::chrono::high_resolution_clock::now();
        
        if (use_cuda && cuda_available) {
            generateFractalCuda();
        } else {
            generateFractalParallel();
        }
        
        auto end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        
        std::string compute_method = use_cuda && cuda_available ? "CUDA GPU" : ("CPU (" + std::to_string(num_threads) + " threads)");
        
        is_generating = false;
    }
    
    void runBenchmark() {
        if (is_generating.load()) return;
        
        benchmark_times.clear();
        
        bool original_use_cuda = use_cuda;
        
        use_cuda = false;
        auto start = std::chrono::high_resolution_clock::now();
        generateFractalSerial();
        auto end = std::chrono::high_resolution_clock::now();
        double serial_time = std::chrono::duration<double, std::milli>(end - start).count();
        benchmark_times.push_back(serial_time);
        
        start = std::chrono::high_resolution_clock::now();
        generateFractalParallel();
        end = std::chrono::high_resolution_clock::now();
        double parallel_time = std::chrono::duration<double, std::milli>(end - start).count();
        benchmark_times.push_back(parallel_time);
        
        if (cuda_available) {
            use_cuda = true;
            start = std::chrono::high_resolution_clock::now();
            generateFractalCuda();
            end = std::chrono::high_resolution_clock::now();
            double cuda_time = std::chrono::duration<double, std::milli>(end - start).count();
            benchmark_times.push_back(cuda_time);
        }
        
        use_cuda = original_use_cuda;
        
        updateTexture();
        updateInfoText();
    }
    
    void updateTexture() {
        texture.update(pixels.data());
        sprite.setTexture(texture);
    }
    
    void zoom(int mouse_x, int mouse_y, double factor) {
        double center_real = min_real + (max_real - min_real) * mouse_x / (fractal_width - 1);
        double center_imag = min_imag + (max_imag - min_imag) * mouse_y / (fractal_height - 1);
        
        double real_range = (max_real - min_real) * factor;
        double imag_range = (max_imag - min_imag) * factor;
        
        min_real = center_real - real_range / 2.0;
        max_real = center_real + real_range / 2.0;
        min_imag = center_imag - imag_range / 2.0;
        max_imag = center_imag + imag_range / 2.0;
    }
    
    void zoomCenter(double factor) {
        double center_real = (min_real + max_real) / 2.0;
        double center_imag = (min_imag + max_imag) / 2.0;
        
        double real_range = (max_real - min_real) * factor;
        double imag_range = (max_imag - min_imag) * factor;
        
        min_real = center_real - real_range / 2.0;
        max_real = center_real + real_range / 2.0;
        min_imag = center_imag - imag_range / 2.0;
        max_imag = center_imag + imag_range / 2.0;
    }
    
    void pan(int dx, int dy) {
        double real_range = max_real - min_real;
        double imag_range = max_imag - min_imag;
        
        double real_delta = -dx * real_range / fractal_width;
        double imag_delta = -dy * imag_range / fractal_height;
        
        min_real += real_delta;
        max_real += real_delta;
        min_imag += imag_delta;
        max_imag += imag_delta;
    }
    
    void resetView() {
        if (julia_mode) {
            min_real = -2.0;
            max_real = 2.0;
            min_imag = -2.0;
            max_imag = 2.0;
        } else {
            min_real = -2.5;
            max_real = 1.0;
            min_imag = -1.25;
            max_imag = 1.25;
        }
    }
    
    void handleButtonClick(int button_index) {
        int offset = cuda_available ? 1 : 0;
        int adjusted_index = button_index - offset;

        switch (button_index) {
            case 0:
                julia_mode = !julia_mode;
                resetView();
                buttons[0].updateText(julia_mode ? "Switch to Mandelbrot" : "Switch to Julia");
                generateFractal();
                updateTexture();
                updateInfoText();
                break;

            case 1:
                use_cuda = false;
                buttons[1].updateText("CPU Serial (Active)");
                buttons[2].updateText("CPU Parallel");
                if (cuda_available) buttons[3].updateText("GPU CUDA");
                generateFractal();
                updateTexture();
                updateInfoText();
                break;

            case 2:
                use_cuda = false;
                buttons[1].updateText("CPU Serial");
                buttons[2].updateText("CPU Parallel (Active)");
                if (cuda_available) buttons[3].updateText("GPU CUDA");
                generateFractal();
                updateTexture();
                updateInfoText();
                break;

            case 3:
                if (cuda_available) {
                    use_cuda = true;
                    buttons[1].updateText("CPU Serial");
                    buttons[2].updateText("CPU Parallel");
                    buttons[3].updateText("GPU CUDA (Active)");
                    generateFractal();
                    updateTexture();
                    updateInfoText();
                }
                break;

            default:
                switch (adjusted_index) {
                    case 3:
                        zoomCenter(0.5);
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                        break;

                    case 4:
                        zoomCenter(2.0);
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                        break;

                    case 5:
                        resetView();
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                        break;

                    case 6:
                        max_iterations += 50;
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                        break;

                    case 7:
                        if (max_iterations > 50) {
                            max_iterations -= 50;
                            generateFractal();
                            updateTexture();
                            updateInfoText();
                        }
                        break;

                    case 8:
                        runBenchmark();
                        break;

                    case 9:
                        if (julia_mode) {
                            julia_c = std::complex<double>(-0.7, 0.27015);
                            generateFractal();
                            updateTexture();
                            updateInfoText();
                        }
                        break;

                    case 10:
                        if (julia_mode) {
                            julia_c = std::complex<double>(-0.8, 0.156);
                            generateFractal();
                            updateTexture();
                            updateInfoText();
                        }
                        break;

                    case 11:
                        if (julia_mode) {
                            julia_c = std::complex<double>(-0.4, 0.6);
                            generateFractal();
                            updateTexture();
                            updateInfoText();
                        }
                        break;
                }
                break;
        }
    }
    
    void handleEvents() {
        sf::Event event;
        while (window.pollEvent(event)) {
            switch (event.type) {
                case sf::Event::Closed:
                    window.close();
                    break;
                    
                case sf::Event::MouseButtonPressed:
                    if (event.mouseButton.button == sf::Mouse::Left) {
                        sf::Vector2i mousePos(event.mouseButton.x, event.mouseButton.y);
                        
                        bool button_clicked = false;
                        for (size_t i = 0; i < buttons.size(); i++) {
                            if (buttons[i].isClicked(mousePos)) {
                                int offset = cuda_available ? 1 : 0;
                                if (i >= 9+offset && i <= 11+offset && !julia_mode) continue;
                                
                                buttons[i].setPressed(true);
                                handleButtonClick(i);
                                button_clicked = true;
                                break;
                            }
                        }
                        
                        if (!button_clicked && event.mouseButton.x < fractal_width) {
                            left_dragging = true;
                            drag_start = mousePos;
                        }
                    }
                    break;
                    
                case sf::Event::MouseButtonReleased:
                    if (event.mouseButton.button == sf::Mouse::Left) {
                        for (auto& button : buttons) {
                            button.setPressed(false);
                        }
                        
                        left_dragging = false;
                    }
                    break;
                    
                case sf::Event::MouseMoved:
                    current_mouse_pos = sf::Vector2i(event.mouseMove.x, event.mouseMove.y);
                    updateCursorText();
                    
                    if (julia_mode && event.mouseMove.x < fractal_width && 
                        !left_dragging &&
                        julia_update_clock.getElapsedTime().asSeconds() > julia_update_interval) {
                        double real = (double)event.mouseMove.x / fractal_width * 4.0 - 2.0;
                        double imag = (double)event.mouseMove.y / fractal_height * 4.0 - 2.0;
                        julia_c = std::complex<double>(real, imag);
                        
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                        julia_update_clock.restart();
                    } else if (left_dragging && event.mouseMove.x < fractal_width) {
                        sf::Vector2i current_pos(event.mouseMove.x, event.mouseMove.y);
                        sf::Vector2i delta = current_pos - drag_start;
                        
                        pan(delta.x, delta.y);
                        drag_start = current_pos;
                        
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                    }
                    break;
                    
                case sf::Event::MouseWheelScrolled:
                    if (event.mouseWheelScroll.x < fractal_width) {
                        if (event.mouseWheelScroll.delta > 0) {
                            zoom(event.mouseWheelScroll.x, event.mouseWheelScroll.y, 0.8);
                        } else {
                            zoom(event.mouseWheelScroll.x, event.mouseWheelScroll.y, 1.25);
                        }
                        generateFractal();
                        updateTexture();
                        updateInfoText();
                    }
                    break;
            }
        }
    }
    
    void run() {
        while (window.isOpen()) {
            handleEvents();
            
            window.clear(sf::Color(40, 40, 40));
            
            window.draw(sprite);
            
            sf::RectangleShape gui_panel(sf::Vector2f(250, window_height));
            gui_panel.setPosition(fractal_width, 0);
            gui_panel.setFillColor(sf::Color(30, 30, 30));
            window.draw(gui_panel);
            
            int offset = cuda_available ? 1 : 0;
            for (size_t i = 0; i < buttons.size(); i++) {
                if (i >= 9+offset && i <= 11+offset && !julia_mode) continue;
                buttons[i].draw(window);
            }
            
            window.draw(info_text);
            
            window.draw(cursor_text);
            
            window.display();
        }
    }
};

int main() {
    int width = 800, height = 600;
    
    std::string input;
    std::getline(std::cin, input);
    
    if (!input.empty()) {
        std::stringstream ss(input);
        ss >> width >> height;
    }
    
    try {
        MandelbrotViewer viewer(width, height);
        viewer.run();
    } catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
        return 1;
    }
    
    return 0;
}